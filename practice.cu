#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>


// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(float *a, float *b, float *c, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.0f;
    for (int i = 0; i < width; ++i) {
        sum += a[row * width + i] * b[i * width + col];
    }

    c[row * width + col] = sum;
}

int main() {
    int width = 1024;
    int size = width * width * sizeof(float);

    // float *h_a = new float[size];
    float *h_b = new float[size];
    float *h_c = new float[size];

// Initialize matrices h_a and h_b with non-zero values
for (int i = 0; i < width * width; ++i) {
    h_a[i] = static_cast<float>(i % width) + 1.0f;
    h_b[i] = static_cast<float>(i % width) + 1.0f;
}


    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    dim3 dimGrid(width / 16, width / 16);
    dim3 dimBlock(16, 16);

    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // After hipMemcpy and kernel launch
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Handle error appropriately, possibly exit the program
    }

	// Print the top-left 4x4 elements of the resulting matrix
	std::cout << "Top-left 4x4 elements of the resulting matrix:" << std::endl;
	for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 4; ++j) {
        std::cout << h_c[i * width + j] << " ";
    }
    std::cout << std::endl;
}


    // Your cleanup code
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
